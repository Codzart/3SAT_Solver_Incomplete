#include "hip/hip_runtime.h"
// Incomplete boolean satisfiability solver
// Data set chosen from ' http://www.cs.ubc.ca/~hoos/SATLIB/benchm.html '
// Data format is DIMACS
// @uthor : Stavan Karia

#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <cmath>
#include <string>
#include <vector>
#include <list>
#include <hash_map>
#include <iostream> 
#include <sstream>
#include "common.h"
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime_api.h>
#include <>

// number of tries to check for random assignment
#define MAX_TRIES 256


// function to assign random values to the literals left in the new formula
// 'formulaGPU' integer array which consists of each literal in the clause
// 'assignment' hash_map to store assignment of literals after simplification 
// 'literalCount' number of literals
// 'formulaSize' size of formula array
// 'clauseCount' number of clauses
// 'seed' seed for Random Number generator
// 'successGPU' decision variable
__global__ void assignRandomValuesGPU(int * formulaGPU,int * assignment,int literalCount,int formulaSize, int *clause, int clauseCount,int seed, int *successGPU){
	int Id=threadIdx.x+blockIdx.x*blockDim.x;
	int lit,trueVal=0,lim=100000;
	// each threads local new assignment array to try for random assignment
	int *assignLocal; 
	assignLocal=new int[literalCount];
	for(int i=1;i<=literalCount;i++){
		assignLocal[i-1]=assignment[i-1];
		if(assignment[i-1]==2){
			// random number generator generator
			long a = 100001;
			a = (a * 125 * (Id+1) * i) % 27962039;
			a = ((a % lim) + 1 + i);
			// give polarity to literal
			// 1 is true
			// 0 is false
			// 2 is unassigned
			if(a<50000){
				assignLocal[i-1]=1;
			}
			else{
				assignLocal[i-1]=0;
			}
		}
	}

	// check for satisfiability of each clause
	// clause[k] is a start and end pointer to the clause in formulaGPU array
	for(int k=0;k<formulaSize-1;k++){
		for(int j=clause[k];j<clause[k+1];j++){
			lit=formulaGPU[j];
			if(lit > 0 && assignLocal[j] == 1){
				trueVal++;
				break;
			}
			else if (lit < 0 && assignLocal[j] == 0){
				trueVal++;
				break;
		
			}
		}
	}

	// if all clauses are satisfiabil, print the solution assignment
	// all possible assignments shall be printed here, by each thread
	if(trueVal==clauseCount){
		printf("A possible solution is : \n");
		printf("Literal\tValue\n");
		for(int l=1;l<=literalCount;l++){
			printf("%d\t%d\n",l,assignLocal[l-1]);
		}
		*successGPU=1;
	}
	else{
		*successGPU=0;
	}

}

// function to perform the boolean satisfiability on the GPU 
void satSolverGPU(){
	int literalCount = 75;													// number of literals 
	bool success=false;														// variable to check for satisfiability of formula
	vector<string> formula;													// vector of strings to store clauses	
	int * assignment;														// int array to store assignment of values to literals 
	assignment = new int[literalCount];
	// read data from file
	formula=readFile();
	// set all literals assignment value as unassigned i.e. 2
	// 1 for true
	// 0 for false
	initAssignmentGPU(assignment,literalCount);	
	cout<<"-------------------------------------------------------------------------------"<<endl;
	cout<<"GPU OUTPUT"<<endl;
	cout<<"-------------------------------------------------------------------------------"<<endl;
	// solve the formula
	success=solver(formula,assignment,literalCount);	
	// display the results on the terminal
	displaySolutionGPU(assignment,literalCount,success);
}

// function to perform unit propagation with
// 'formula' : vector of strings which consists of clauses
// 'assignment' is the assignment given to literals namely, true i.e. 1  or false i.e. 0 or unassigned i.e. 2
vector<string> unitPropagationGPU(vector<string> formula,int * assignment){
	// delimiter used to show logical OR 
	string delimiter=" ";
	// position of the delimiter found
	size_t position=0;
	// literal which is to be deleted as it is a unit clause
	int literalChosen,i=0;
	// variable to traverse accross the formula
	do{
		if(formula[i].length()!=0){
			position=formula[i].find(delimiter);	
			// if there is no delimiter it returns 'npos'
			// if it returns npos, that means it is a unit clause
			// assign true if polarity of the literal is positive else assign false
			if(position==string::npos){
				stringstream convert(formula[i]);
				convert>>literalChosen;
				// assign boolean value true or false depending on the polarity of the variable in the unit clause
				if(literalChosen>0){
					assignment[literalChosen-1]=1;
				}
				else if (literalChosen<0){
					assignment[(-1*literalChosen)-1]=0;
				}
				// delete clauses with the literal with same polarity
				// and remove same literal with opposite polarity from all clauses 
				formula=deleteClausesGPU(formula,literalChosen);
				// reset search from first element	
				i=0;
			}
			else{
					i++;
					if(i==formula.size()){
						return formula;
					}
				}
		}
		else{
			i=formula.size();
			break;
		}
		if(formula.size()==0){
			return formula;
		}
	}while(i<formula.size());	
	return formula;
}

// function to solve the formula
bool solver(vector<string> formula,int * assignment,int literalCount){
	// decision variables to continue cycle of unit propgation
	int *goAhead=new int;												
	*goAhead=1;
	int seed;
	bool success= false;	
	do{
		// perform unit propagation
		formula=unitPropagationGPU(formula, assignment);
		if(formula.size()==0){
			*goAhead=1;
			return true;
		}
		// perform pure literal assignment
		formula=pureLiteralAssignmentGPU(formula,goAhead,literalCount,assignment);

		// peform unit propagation if any more clauses added in pureLiteralAssignment 
	}while(*goAhead);

	// generate a random seed
	seed=rand();

	//number of elements in the formula
	int formulaElems=0; 

	// delimiter in the clauses
	string delimiter=" ";

	// find number of elements
	for(int i=0;i<formula.size();i++){
		size_t pos = 0;
		string token;
		string temp=formula[i];
		pos = temp.find(delimiter);
		while (!(pos == string::npos)) {
			token = temp.substr(0, pos);
			formulaElems++;
			temp.erase(0, pos + delimiter.length());
			pos = temp.find(delimiter);
			if(pos==string::npos && temp.length()!=0){
				pos=temp.length();	
			}
		}
	}

	// create array of formula elements as integers
	// create pointers to clause start and end locatins in formula array
	int * formulaArray,*clause;
	formulaArray = new int[formulaElems];
	clause = new int[formula.size()]; 
	int literalChosen,counter=0;
	for(int i=0;i<formula.size();i++){
		size_t pos = 0;
		string token;
		string temp=formula[i];
		pos = temp.find(delimiter);
		clause[i]=counter;
		while (!(pos == string::npos)) {
			token = temp.substr(0, pos);
			stringstream convert(token);
			convert>>literalChosen;		
			formulaArray[counter]=literalChosen;
			counter++;
			temp.erase(0, pos + delimiter.length());
			pos = temp.find(delimiter);
			if(pos==string::npos && temp.length()!=0){
				pos=temp.length();	
			}
		}
	}
	
	// create variables for GPU
	int *formulaGPU;
	int *clauseGPU;
	int *assignmentGPU;
	int *successGPU;
	// allocate memory on GPU for above mentioned variables
	hipMalloc((void**)&formulaGPU, formulaElems*sizeof(int));
	hipError_t	status = hipGetLastError();
		if (status != hipSuccess) 
		{
			std::cout << "Kernel failed 1: " << hipGetErrorString(status) << std::endl;
			return false;
		}

	hipMalloc((void**)&assignmentGPU, literalCount*sizeof(int));
	status = hipGetLastError();
		if (status != hipSuccess) 
		{
			std::cout << "Kernel failed 2: " << hipGetErrorString(status) << std::endl;
			return false;
		}

	hipMalloc((void**)&clauseGPU, formula.size()*sizeof(int));
		status = hipGetLastError();
		if (status != hipSuccess) 
		{
			std::cout << "Kernel failed 3: " << hipGetErrorString(status) << std::endl;
			return false;
		}
	hipMalloc((void**)&successGPU, sizeof(int));
	status = hipGetLastError();
		if (status != hipSuccess) 
		{
			std::cout << "Kernel failed 4: " << hipGetErrorString(status) << std::endl;
			return false;
		}
	
	// copy data from CPU variables to GPU variables
	hipMemcpy(formulaGPU, formulaArray,formulaElems*sizeof(int),hipMemcpyHostToDevice);
	status = hipGetLastError();
		if (status != hipSuccess) 
		{
			std::cout << "Kernel failed 5: " << hipGetErrorString(status) << std::endl;
			return false;
		}

	hipMemcpy(assignmentGPU,assignment,literalCount*sizeof(int),hipMemcpyHostToDevice);
	status = hipGetLastError();
		if (status != hipSuccess) 
		{
			std::cout << "Kernel failed 6: " << hipGetErrorString(status) << std::endl;
			return false;
		}


	hipMemcpy(clauseGPU,clause,formula.size()*sizeof(int),hipMemcpyHostToDevice);
	status = hipGetLastError();
		if (status != hipSuccess) 
		{
			std::cout << "Kernel failed 7: " << hipGetErrorString(status) << std::endl;
			return false;
		}

	// give block and grid dimensions for GPU kernel
	// 1D GRID 1D block
	dim3 dimBlock(256,1);
	status = hipGetLastError();
		if (status != hipSuccess) 
		{
			std::cout << "Kernel failed 8: " << hipGetErrorString(status) << std::endl;
			return false;
		}

	int gridX=1;//MAX_TRIES/1024;
	dim3 dimGrid(gridX,1);
		status = hipGetLastError();
		if (status != hipSuccess) 
		{
			std::cout << "Kernel failed 9: " << hipGetErrorString(status) << std::endl;
			return false;
		}
	// kernel to assign random values on each thread and check for that threads satisfiability
	assignRandomValuesGPU<<<dimGrid,dimBlock>>>(formulaGPU,assignmentGPU,literalCount,formulaElems,clauseGPU,formula.size(),seed,successGPU);
		status = hipGetLastError();
		if (status != hipSuccess) 
		{
			std::cout << "Kernel failed 10: " << hipGetErrorString(status) << std::endl;
			return false;
		}

	if(success==true){
		return true;
	}
	return success;
}




// function to delete clauses with 
// 'literalChosen' as one of its literals
vector<string> deleteClausesGPU(vector<string> formula,int literalChosen){
	// position of literal in formula
	size_t position=0;
	size_t positionOpp=0;
	for(int i=0;i<formula.size();i++){
			// check if chosen literal is found
			ostringstream oss;
			oss<<literalChosen;
			position=formula[i].find(oss.str());
			// check if chosen literal's opposite polarity is found
			ostringstream ossOpp;
			ossOpp<<(literalChosen*-1);
			positionOpp=formula[i].find(ossOpp.str());
			// if the literal is present in the clause
			// then delete / erase the clause
			if(position!=string::npos){
				//cout<<"deleted clause "<<formula[i]<<endl;
				formula.erase(formula.begin()+i);
				i--;
			}
			// if literal with opposite polarity is found
			// remove the literal from the clause
			else if(positionOpp!=string::npos){
				// removes the first occurence of the literal with opposite polarity
				// if the literal is the first in the clause
				if(positionOpp==0){
					formula[i].erase(positionOpp,ossOpp.str().length()+1);
				}
				// if the literal is at the end of the clause
				else if (positionOpp==formula[i].length()-ossOpp.str().length()){
					formula[i].erase(positionOpp-1,ossOpp.str().length()+1);
				}
				// if the literal is somewhere in the middle section of the clause
				else{
					formula[i].erase(positionOpp,ossOpp.str().length()+1);
				}
				// removes all occurences after the first of the literal with opposite polarity
				while((positionOpp=formula[i].find(ossOpp.str()))!=string::npos){
					// if the literal is the first in the clause
					if(positionOpp==0){
						formula[i].erase(positionOpp,ossOpp.str().length()+1);
					}
					// if the literal is at the end of the clause
					else if (positionOpp==formula[i].length()-ossOpp.str().length()){
						formula[i].erase(positionOpp-1,ossOpp.str().length()+1);			
					}
					// if the literal is somwhere in the middle section of the clause
					else{
						formula[i].erase(positionOpp,ossOpp.str().length()+1);
					}
				}
		}
	}
	return formula;
}


// function to identify pure literals and remove them from clauses, then add a new unit clause for theses literals
// 'formula' vector of strings which consists of clauses
// 'goAhead' decision variable for looping
// 'literalCount' number of literals
// 'assignment' integer array to store assignment of literals after simplification
vector<string> pureLiteralAssignmentGPU(vector<string> formula, int * goAhead, int literalCount, int * assignment){
	*goAhead=0;
	int positive, neg, n;
	size_t pos,posOpp;
	string str1,str2;
	// for each literal and its polarity
	// check all clauses 
	for(int i=1;i<=literalCount;i++){
		if(assignment[i-1]==2){
			positive=0;
			neg=0;
			n=0;
			for(int j=0;j<formula.size();j++){
				ostringstream oss;
				oss<<i;
				ostringstream ossOpp;
				ossOpp<<i*(-1);
				pos=formula[j].find(oss.str());
				posOpp=formula[j].find(ossOpp.str());
				str1=oss.str();
				str2=ossOpp.str();
				// if positive polarity found
				if (pos!=string::npos){
					positive++; 
					n++;
		     	}
				// if negative polarity found
				if (posOpp!=string::npos){
					neg++; 
       				n++;
       			}
			}      
			// if literal found only in positive polarity
			 if (n!=0 && positive!=0 && positive == n){
				assignment[i-1] = 1; 
				// add literal to clause
				formula.push_back(str1);
				*goAhead=1;
			 }
   			// if literal found only in negative polarity
			else if (n!=0 && neg!=0 && neg == n){
				assignment[i-1] = 0; 
				// add literal to clause
				formula.push_back(str2);
				*goAhead=1;
			}
		}
	}
	return formula;
}

// display the solution assignment for the given formula
void displaySolutionGPU(int *assignment,int literalCount,bool success){
	if(success==1){
		cout<<"Congratulations !"<<endl;
		cout<<endl<<"Formula Satisfied !"<<endl;
		cout<<"( All the lierals not assigned any value below are to be assumed as false )"<<endl;  
		cout<<"Assignments are : "<<endl;
		cout<<"-----------------------------------"<<endl;
		cout<<"Literal \t"<<"Value"<<endl;
		for(int i=1;i<=literalCount;i++){
			printf("%d\t%d\n",i,assignment[i-1]);
		}
	}
	else{
		cout<<"Sorry :("<<endl;
		cout<<"There is no solution i.e. possible assignment for this formula !"<<endl;
	}

}


void initAssignmentGPU(int * assignment,int literalCount){
	for(int i=1;i<=literalCount;i++){
		assignment[i-1]=2;
	}
}
